#include "hip/hip_runtime.h"
#include "JointTreeToPointSet.h"

#include "Matrix/MatrixFunc.h"
#include "Topology/NeighborPointQuery.h"

#include <iostream>

namespace dyno
{
	template<typename TDataType>
	JointTreeToPointSet<TDataType>::JointTreeToPointSet()
		: TopologyMapping()
	{

	}


	template<typename TDataType>
	void JointTreeToPointSet<TDataType>::set(
			std::shared_ptr<PointSet<TDataType>> from,
			std::vector<std::shared_ptr<Cluster<TDataType>>>* clusters,
			std::vector<std::shared_ptr<JointTree<TDataType>>>* jointTree)
	{
		m_from = from;
		m_clusters = clusters;
		m_jointTree = jointTree;

	}

	template<typename TDataType>
	JointTreeToPointSet<TDataType>::~JointTreeToPointSet()
	{

	}
	// template<typename TDataType>
	// void JointTreeToPointSet<TDataType>:: UpdateAnimationFor(
	// 	vector<int> indices,
	// 	vector<Real> weights,
	// 	Mat4f Mt,
	// 	Mat4f Mtl,
	// 	Mat4f GlobalTransform,
	// 	int size2d,
	// 	int Size3d,
	// 	vector<Coord> old_points,
	// 	vector<Coord> new_points)
	// {
	// 	// int pId = threadIdx.x + (blockIdx.x * blockDim.x);
	// 	// if (pId >= indices.size()) return;	
	// 	for (int pId = 0; pId < indices.size(); pId++){
	// 		// 对于读入文件的特殊处理
	// 		int indexConvert = 0;
	// 		if(indices[pId] >= size2d + Size3d) return;
	// 		if(indices[pId] < Size3d) indexConvert = indices[pId] + size2d;
	// 		else indexConvert = indices[pId] - Size3d;

	// 		Coord old_p = old_points[indexConvert];

	// 		// TODO:Update Coord
	// 		Vec4f tmp_p(old_p[0], old_p[1], old_p[2], 1);
			
	// 		// ?
	// 		tmp_p = GlobalTransform * Mtl * tmp_p;
	// 		old_p[0] = tmp_p[0] / tmp_p[4];
	// 		old_p[1] = tmp_p[1] / tmp_p[4];
	// 		old_p[2] = tmp_p[2] / tmp_p[4];

	// 		new_points[indexConvert] += weights[pId] * old_p;
	// 	}
	// }

	// 对每个关节所控制的点集做动画更新
	template <typename Real, typename Coord>
	__global__ void UpdateAnimation(
		DArray<int> indices,
		DArray<Real> weights,
		Mat4f Mt,
		Mat4f Mtl,
		Mat4f GlobalTransform,
		int size2d,
		int Size3d,
		DArray<Coord> points)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= indices.size()) return;	
		
		// 对于读入文件的特殊处理
		int indexConvert = 0;
		if(indices[pId] >= size2d + Size3d) return;
		if(indices[pId] < Size3d) indexConvert = indices[pId] + size2d;
		else indexConvert = indices[pId] - Size3d;

		Coord old_p = points[indexConvert];

		// TODO:Update Coord
		Vec4f tmp_p(old_p[0], old_p[1], old_p[2], 1);
		
		// ?
		tmp_p = GlobalTransform * Mtl * tmp_p;
		old_p[0] = tmp_p[0] / tmp_p[4];
		old_p[1] = tmp_p[1] / tmp_p[4];
		old_p[2] = tmp_p[2] / tmp_p[4];

		points[indexConvert] = old_p;
	}

	// 初始化每个点所受控制的关节关系
	template <typename Real>
	__global__ void InitMatch(
		DArray<int> indices,
		DArray<Real> weights,
		DArrayList<int> clusterIds,
		DArrayList<Real> clusterWeights,
		int clusterId) 
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= indices.size()) return;
		int pointId = indices[pId];
		clusterWeights[pointId].atomicInsert(weights[pId]);
		clusterIds[pointId].atomicInsert(clusterId);
	}

	template<typename TDataType>
	bool JointTreeToPointSet<TDataType>::initializeImpl()
	{
		match();
		return true;
	}

	template<typename TDataType>
	bool JointTreeToPointSet<TDataType>::apply()
	{
		for (auto joint : *m_jointTree)
		{
			joint->getGlobalTransform(); // DFS 遍历
		}

		std::shared_ptr<Cluster<TDataType>> v;
		for (int i = 0; i < this->m_clusters->size(); i++)
		{
			v = (*(this->m_clusters))[i];
			uint pDim = v->m_indices.size();
			
			cuExecute(pDim,
				UpdateAnimation,
				v->m_indices,
				v->m_weights,
				v->m_transform,
				v->m_transformLink,
				(*m_jointTree)[v->m_jointIndex]->GlobalTransform,
				// Mat4f(0.0),
				3561,	
				43300,
				m_from->getPoints());
			cuSynchronize();

			// UpdateAnimationFor(
			// v->m_indices,
			// v->m_weights,
			// v->m_transform,
			// v->m_transformLink,
			// (*m_jointTree)[v->m_jointIndex]->GlobalTransform,
			// 3561,	
			// 43300,
			// m_from->getPoints());
		}


		// for (auto v : *(this->m_clusters))
		// {
		// 	uint pDim = v->m_indices.size();
		// 	cuExecute(pDim,
		// 		UpdateAnimation,
		// 		v->m_indices,
		// 		v->m_weights,
		// 		v->m_transform,
		// 		v->m_transformLink,
		// 		(*m_jointTree)[v->m_jointIndex]->getGlobalTransform(),
		// 		this->m_from->getPoints());
		// 	cuSynchronize();
		// }
		return true;
	}



	template<typename TDataType>
	void JointTreeToPointSet<TDataType>::match()
	{
	}

	DEFINE_CLASS(JointTreeToPointSet);
}