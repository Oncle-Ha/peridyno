#include "hip/hip_runtime.h"
#include "NeighborPointQueryJoint.h"
#include "Topology/GridHash.h"
#include <thrust/sort.h>


#define PT_d(s, x, y) printf("%s: %d  pId: %d\n", s, x, y)
#define PT_f(s, x, y) printf("%s: %f  pId: %d\n", s, x, y)
#define PT_e(s, y) printf("[%s]  pId: %d\n", s, y)
namespace dyno
{
	__constant__ int offset_nq2[27][3] = { 
		0, 0, 0,
		0, 0, 1,
		0, 1, 0,
		1, 0, 0,
		0, 0, -1,
		0, -1, 0,
		-1, 0, 0,
		0, 1, 1,
		0, 1, -1,
		0, -1, 1,
		0, -1, -1,
		1, 0, 1,
		1, 0, -1,
		-1, 0, 1,
		-1, 0, -1,
		1, 1, 0,
		1, -1, 0,
		-1, 1, 0,
		-1, -1, 0,
		1, 1, 1,
		1, 1, -1,
		1, -1, 1,
		-1, 1, 1,
		1, -1, -1,
		-1, 1, -1,
		-1, -1, 1,
		-1, -1, -1
	};

	IMPLEMENT_TCLASS(NeighborPointQueryJoint, TDataType)

	template<typename TDataType>
	NeighborPointQueryJoint<TDataType>::NeighborPointQueryJoint()
		: ComputeModule()
	{
		//this->inOther()->tagOptional(true);
	}

	template<typename TDataType>
	NeighborPointQueryJoint<TDataType>::~NeighborPointQueryJoint()
	{
	}

	// max_cap {O(Gird*Point)}
	// count<顶点, 距离, 关节>对
	template<typename Coord, typename JCapsule, typename TDataType>
	__global__ void K_CountNeighbor(
		DArray<Coord> position, 
		GridHash<TDataType> hash, 
		Real h,
		DArray<JCapsule> caps,
		DArray<int> count)  // cap:[<>..]
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= caps.size()) return;
		
		count[pId] = 0;
		JCapsule cap = caps[pId];

		// 削短胶囊体
		Coord tmp_s = (cap.v1 - cap.v0);
		Real tmp_d = tmp_s.norm();
		Real rate =  (h / tmp_d) * 0;
		if(rate > 0.5) rate = 0.5;
		cap.v0 += rate* tmp_s;
		cap.v1 -= rate * tmp_s;

		int3 vId0 = hash.getIndex3(cap.v0);
		int3 vId1 = hash.getIndex3(cap.v1);
		float eps = 1e-6;

		//DEBUG 
		// printf("[(%d,%d,%d)->(%d,%d,%d)] pId: %d\n", vId0.x, vId0.y, vId0.z, vId1.x, vId1.y, vId1.z, pId);

		Coord m = cap.v0;
		Coord s = (cap.v1 - cap.v0);
		Real d = s.norm();

		// 遍历线段所覆盖的Grid
		int3 vId = vId0;
		
		// 线段与立方体求交
		Coord min_v = hash.getMin3(vId);
		Coord max_v = hash.getMax3(vId);
		Coord time1 = (min_v - m) / s;
		Coord time2 = (max_v - m) / s;
		Coord max_t(max(time1[0],time2[0]), max(time1[1],time2[1]), max(time1[2],time2[2]));
		float next_t = min(max_t[0], min(max_t[1], max_t[2]));	//边界时间段	
		// PT_f("time", next_t, pId);
		while(true)
		{	
			int gId = hash.getIndex(vId.x, vId.y, vId.z);

			//DEBUG
			// PT_d("Gird", gId, pId);
			// printf("Gird:%d [%d,%d,%d] pId: %d\n", gId, vId.x, vId.y, vId.z, pId);

			if (gId == -1) break;

			//DEBUG 
			//PT_d("Num", totalNum, pId);
			
			for (int c = 0; c < 27; c++)
			{
				int3 cId;
				cId.x = vId.x + offset_nq2[c][0];
				cId.y = vId.y + offset_nq2[c][1];
				cId.z = vId.z + offset_nq2[c][2];
				if (cId.x >= 0 && cId.y >= 0 && cId.z >= 0) 
				{ 	
					int cNumId = hash.getIndex(cId.x, cId.y, cId.z);
					int totalNum = hash.getCounter(cNumId);
					for (int i = 0; i < totalNum; i++) {
						int nbId = hash.getParticleId(cNumId, i);
						Coord pos_i = position[nbId];
						
						Real d_v0 = (pos_i - cap.v0).norm();
						Real d_v1 = (pos_i - cap.v1).norm();
						Real d_t = (pos_i - m).dot(s);
						Real d_line = fabs(((pos_i - m).cross(s)).norm() / d);
						Real min_d = (d_t < d * d && d_t > 0)? d_line : min(d_v0, d_v1);

						if (min_d < h)
						{
							// PT_f("d_line", d_line, pId);
							count[pId] +=1;
						}
					}
				}
			}
			if (next_t > 1) break; //终点格子

			float tmp_t = -1;
			int3 next_c;	
			

			// 选取线段上最近Grid
			for (int c = 1; c < 27; c++)
			{
				int3 cId;
				cId.x = vId.x + offset_nq2[c][0];
				cId.y = vId.y + offset_nq2[c][1];
				cId.z = vId.z + offset_nq2[c][2];
				if (cId.x >= 0 && cId.y >= 0 && cId.z >= 0) 
				{ 	
					// 线段与立方体求交
					Coord min_v = hash.getMin3(cId);
					Coord max_v = hash.getMax3(cId);
					Coord time1 = (min_v - m) / s;
					Coord time2 = (max_v - m) / s;
					Coord min_t(min(time1[0],time2[0]), min(time1[1],time2[1]), min(time1[2],time2[2]));
					Coord max_t(max(time1[0],time2[0]), max(time1[1],time2[1]), max(time1[2],time2[2]));
					float mint = max(min_t[0], max(min_t[1], min_t[2]));
					float maxt = min(max_t[0], min(max_t[1], max_t[2]));

					// printf("GirdTest [%d,%d,%d] - (%f, %f) < (%f) pId: %d\n", cId.x, cId.y, cId.z, mint, maxt, tmp_t, pId);
					// printf("Checsk( %d )\n", (mint > next_t - eps));
					if (mint > 0 && mint < 1 && mint < maxt && (mint < tmp_t || tmp_t < 0) && mint > next_t - eps)
					{
						tmp_t = maxt;
						next_c = cId;
						//DEBUG
						//if (cId == vId) printf("Error cId == vId\n");
					}
				}
			}
			
			next_t = tmp_t;
			// PT_f("time", next_t, pId);
			if (next_t < 0) break;
			vId = next_c;
		}
		//DEBUG
		PT_d("Count", count[pId], pId);
	}

	// max_cap {O(Gird*Point)} TODO更新同Count一样
	// 寻找关节所延伸胶囊体控制的顶点
	template<typename Coord, typename JCapsule, typename TDataType, typename Pair4f>
	__global__ void K_ComputeNeighbor(
		DArray<Coord> position, 
		GridHash<TDataType> hash, 
		Real h,
		DArray<JCapsule> caps,
		DArray<Pair4f> capPairs,
		DArray<int> count)  // cap:[<>..]
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= caps.size()) return;
		
		JCapsule cap = caps[pId];
		// 削短胶囊体
		Coord tmp_s = (cap.v1 - cap.v0);
		Real tmp_d = tmp_s.norm();
		Real rate =  (h / tmp_d) * 0;
		if(rate > 0.5) rate = 0.5;
		cap.v0 += rate* tmp_s;
		cap.v1 -= rate * tmp_s;

		int3 vId0 = hash.getIndex3(cap.v0);
		int3 vId1 = hash.getIndex3(cap.v1);
		float eps = 1e-6;

		//DEBUG 
		// printf("[(%d,%d,%d)->(%d,%d,%d)] pId:%d\n", vId0.x, vId0.y, vId0.z, vId1.x, vId1.y, vId1.z, pId);

		Coord m = cap.v0;
		Coord s = (cap.v1 - cap.v0);
		Real d = s.norm();
		

		// 遍历线段所覆盖的Grid
		int3 vId = vId0;
		
		// 线段与立方体求交
		// 线段与立方体求交
		Coord min_v = hash.getMin3(vId);
		Coord max_v = hash.getMax3(vId);
		Coord time1 = (min_v - m) / s;
		Coord time2 = (max_v - m) / s;
		Coord max_t(max(time1[0],time2[0]), max(time1[1],time2[1]), max(time1[2],time2[2]));
		float next_t = min(max_t[0], min(max_t[1], max_t[2]));	//边界时间段
		int start = count[pId];
		int cnt = 0;
		while(true)
		{	
			int gId = hash.getIndex(vId.x, vId.y, vId.z);

			//DEBUG
			// PT_d("Gird", gId, pId);

			if (gId == -1) break;
			
			for (int c = 0; c < 27; c++)
			{
				int3 cId;
				cId.x = vId.x + offset_nq2[c][0];
				cId.y = vId.y + offset_nq2[c][1];
				cId.z = vId.z + offset_nq2[c][2];
				if (cId.x >= 0 && cId.y >= 0 && cId.z >= 0) 
				{ 	
					int cNumId = hash.getIndex(cId.x, cId.y, cId.z);
					int totalNum = hash.getCounter(cNumId);
					for (int i = 0; i < totalNum; i++) {
						int nbId = hash.getParticleId(cNumId, i);
						Coord pos_i = position[nbId];

						Real d_v0 = (pos_i - cap.v0).norm();
						Real d_v1 = (pos_i - cap.v1).norm();
						Real d_t = (pos_i - m).dot(s);
						Real d_line = fabs(((pos_i - m).cross(s)).norm() / d);
						Real min_d = (d_t < d * d && d_t > 0)? d_line : min(d_v0, d_v1);

						if (min_d < h)
						{
							// PT_f("MIN", min_d, pId);
							// printf("<id:%d dis:%f joint:%d>  pId:%d\n", nbId, min_d, cap.id_joint, pId);
							capPairs[cnt + start] = (Pair4f(nbId, -min_d, cap.id_joint, cap.id_cap));
							// PT_d("index", cnt + start, pId);
							cnt++;
						}
					}
				}
			}

			if (next_t > 1) break; //终点格子

			float tmp_t = -1;
			int3 next_c;	
			// 选取线段上最近Grid
			for (int c = 1; c < 27; c++)
			{
				int3 cId;
				cId.x = vId.x + offset_nq2[c][0];
				cId.y = vId.y + offset_nq2[c][1];
				cId.z = vId.z + offset_nq2[c][2];
				if (cId.x >= 0 && cId.y >= 0 && cId.z >= 0) 
				{ 	
					// 线段与立方体求交
					Coord min_v = hash.getMin3(cId);
					Coord max_v = hash.getMax3(cId);
					Coord time1 = (min_v - m) / s;
					Coord time2 = (max_v - m) / s;
					Coord min_t(min(time1[0],time2[0]), min(time1[1],time2[1]), min(time1[2],time2[2]));
					Coord max_t(max(time1[0],time2[0]), max(time1[1],time2[1]), max(time1[2],time2[2]));
					float mint = max(min_t[0], max(min_t[1], min_t[2]));
					float maxt = min(max_t[0], min(max_t[1], max_t[2]));
					if (mint > 0 && mint < 1 && mint < maxt && (mint < tmp_t || tmp_t < 0) && mint > next_t - eps)
					{
						tmp_t = maxt;
						next_c = cId;
						//DEBUG
						//if (cId == vId) printf("Error cId == vId\n");
					}
				}
			}

			next_t = tmp_t;
			// PT_f("time", next_t, pId);
			if (next_t < 0) break;
			vId = next_c;
		}
	}


	// max_cap 
	// 统计顶点数
	template<typename Pair4f>
	__global__ void K_CountPoint(
		DArray<Pair4f> capPairs,
		DArray<int> count)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= capPairs.size()) return;
		if( pId == capPairs.size() - 1 || int(capPairs[pId][0]) != int(capPairs[pId + 1][0]))
			count[pId] = 1;
		else 
		{
			// printf("[%d, %d] pId:%d\n", int(capPairs[pId][0]), int(capPairs[pId + 1][0]), pId);
			count[pId] = 0;
		}
		
	}

	// max_point
	// set out<关节, 顶点, 胶囊体>
	template<typename Pair4f, typename Pair3>
	__global__ void K_SetOutPair(
		DArray<Pair4f> capPairs,
		DArray<int> count,
		DArray<Pair3> outPairs)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= capPairs.size()) return;
		if( pId == capPairs.size() - 1 || int(capPairs[pId][0]) != int(capPairs[pId + 1][0]))
		{
			outPairs[count[pId]] = Pair3(capPairs[pId][2], capPairs[pId][0], capPairs[pId][3]);
			// printf("<joint:%d, point:%d>\n", outPairs[count[pId]][0], outPairs[count[pId]][1]);
		}
			
	}

	template<typename TDataType>
	void NeighborPointQueryJoint<TDataType>::compute()
	{
		// Prepare inputs
		auto& points	= this->inPosition()->getData();
		auto& capsules  = this->inCapsule()->getData();
		auto h			= this->inRadius()->getData();

		// Prepare outputs
		if (this->outPJPair()->isEmpty())
			this->outPJPair()->allocate();

		auto& outPairs = this->outPJPair()->getData();

		// uint numJt  = this->inJointSize()->getData();
		uint numPt  = this->inPosition()->getDataPtr()->size();
		uint numCp  = this->inCapsule()->getDataPtr()->size();
		uint sizeLimit = this->varSizeLimit()->getData();

		// Construct hash grid
		Reduction<Coord> reduce;
		Coord hiBound = reduce.maximum(points.begin(), points.size());
		Coord loBound = reduce.minimum(points.begin(), points.size());

		GridHash<TDataType> hashGrid;
		hashGrid.setSpace(h, loBound - Coord(h), hiBound + Coord(h));
		hashGrid.clear();
		hashGrid.construct(points);

		DArray<int> count;
		
		count.resize(numCp);
		cuExecute(numCp,
			K_CountNeighbor,
			points,
			hashGrid,
			h,
			capsules,
			count);
		cuSynchronize();

		int numPair = m_reduce.accumulate(count.begin(), count.size());
		m_scan.exclusive(count, true);

		//FixMe: numPair = 0?

		DArray<Pair4f>capJointPairs;
		capJointPairs.resize(numPair);
		cuExecute(numCp,
			K_ComputeNeighbor,
			points,
			hashGrid,
			h,
			capsules,
			capJointPairs,
			count);
		cuSynchronize();

		thrust::sort(thrust::device, capJointPairs.begin(), capJointPairs.begin() + capJointPairs.size());
		
		count.resize(numPair);
		cuExecute(numPair,
			K_CountPoint,
			capJointPairs,
			count);
		cuSynchronize();

		int numPoint = m_reduce.accumulate(count.begin(), count.size());
		m_scan.exclusive(count, true);

		assert(numPoint >= 0);

		outPairs.resize(numPoint);
		cuExecute(numPair,
			K_SetOutPair,
			capJointPairs,
			count,
			outPairs);
		cuSynchronize();

		capJointPairs.clear();
		count.clear();
		hashGrid.clear();
	}

	DEFINE_CLASS(NeighborPointQueryJoint);
}