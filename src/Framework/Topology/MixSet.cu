#include "hip/hip_runtime.h"
#include "MixSet.h"

#include <string>
#include <sstream>

#include <thrust/sort.h>

namespace dyno
{
    IMPLEMENT_CLASS_1(MixSet, TDataType)

    template<typename TDataType>
    MixSet<TDataType>::MixSet()
    {

    }
    template<typename TDataType>
    MixSet<TDataType>::~MixSet()
    {

    }

	template<typename TDataType>
	void MixSet<TDataType>::setTriPoints(std::vector<Coord>& pos)
	{
		//printf("%d\n", pos.size());
        m_triPointSize = pos.size();
		m_coords.resize(pos.size());
		m_coords.assign(pos);

		tagAsChanged();
	}

	template<typename TDataType>
	void MixSet<TDataType>::setTetPoints(std::vector<Coord>& pos)
	{
        assert(m_triPointSize > 0);

        DArray<Coord> tmp;
        tmp.resize(m_triPointSize);
        tmp.assign(m_coords);

        m_tetPointSize = pos.size();
		m_coords.resize(m_triPointSize + m_tetPointSize);
		m_coords.assign(tmp, m_triPointSize, 0, 0);
		//FIXME bug: assign(vector)
        // m_coords.assign(pos, m_tetPointSize, m_triPointSize, 0); 

        tmp.resize(m_tetPointSize);
        tmp.assign(pos);	
		m_coords.assign(tmp, m_tetPointSize, m_triPointSize, 0);

		tagAsChanged();
	}

	template<typename TDataType>
	void MixSet<TDataType>::setTriangles(std::vector<Triangle>& triangles)
	{
		m_triangles.resize(triangles.size());
		m_triangles.assign(triangles);
	}

	template<typename TDataType>
	void MixSet<TDataType>::setTetrahedrons(std::vector<Tetrahedron>& tetrahedrons)
	{
		m_tethedrons.resize(tetrahedrons.size());
		m_tethedrons.assign(tetrahedrons);
	}

    template<typename TDataType>
	void MixSet<TDataType>::loadObjFile(std::string filename)
	{
		if (filename.size() < 5 || filename.substr(filename.size() - 4) != std::string(".obj")) {
			std::cerr << "Error: Expected OBJ file with filename of the form <name>.obj.\n";
			exit(-1);
		}

		std::ifstream infile(filename);
		if (!infile) {
			std::cerr << "Failed to open. Terminating.\n";
			exit(-1);
		}

		int ignored_lines = 0;
		std::string line;
		std::vector<Coord> vertList;
		std::vector<Triangle> faceList;
		while (!infile.eof()) {
			std::getline(infile, line);

			//.obj files sometimes contain vertex normals indicated by "vn"
			if (line.substr(0, 1) == std::string("v") && line.substr(0, 2) != std::string("vn")) {
				std::stringstream data(line);
				char c;
				Coord point;
				data >> c >> point[0] >> point[1] >> point[2];
				vertList.push_back(point);
			}
			else if (line.substr(0, 1) == std::string("f")) {
				std::stringstream data(line);
				char c;
				int v0, v1, v2;
				data >> c >> v0 >> v1 >> v2;
				faceList.push_back(Triangle(v0 - 1, v1 - 1, v2 - 1));
			}
			else {
				++ignored_lines;
			}
		}
		infile.close();

		setTriPoints(vertList);
		setTriangles(faceList);
	}
            

    template<typename TDataType>
	void MixSet<TDataType>::loadTetFile(std::string filename)
	{
		std::string filename_node = filename;	filename_node.append(".node");
		std::string filename_ele = filename;	filename_ele.append(".ele");

		std::ifstream infile_node(filename_node);
		std::ifstream infile_ele(filename_ele);
		if (!infile_node || !infile_ele) {
			std::cerr << "Failed to open the tetrahedron file. Terminating.\n";
			exit(-1);
		}

		std::string line;
		std::getline(infile_node, line);
		std::stringstream ss_node(line);

		int node_num;
		ss_node >> node_num;
		std::vector<Coord> nodes;
		for (int i = 0; i < node_num; i++)
		{
			std::getline(infile_node, line);
			std::stringstream data(line);
			int id;
			Coord v;
			data >> id >> v[0] >> v[1] >> v[2];
			nodes.push_back(v);
		}

		
		std::getline(infile_ele, line);
		std::stringstream ss_ele(line);

		int ele_num;
		ss_ele >> ele_num;
		std::vector<Tetrahedron> tets;
		for (int i = 0; i < ele_num; i++)
		{
			std::getline(infile_ele, line);
			std::stringstream data(line);
			int id;
			Tetrahedron tet;
			data >> id >> tet[0] >> tet[1] >> tet[2] >> tet[3];
			tet[0] -= 1;
			tet[1] -= 1;
			tet[2] -= 1;
			tet[3] -= 1;
			tets.push_back(tet);
		}

		setTetPoints(nodes);
        setTetrahedrons(tets);
	}


    template<typename TDataType>
    void MixSet<TDataType>::loadMixFile(std::string filename)
    {
        std::string filename_2d = filename; filename_2d.append("_2d.obj");
        std::string filename_3d = filename; filename_3d.append("_3d");
        loadObjFile(filename_2d);
        loadTetFile(filename_3d);

        this->getJointVer();
    }

    template<typename Coord, typename FKey>
    __global__ void MS_SetupKeys(
        DArray<int> ids,
        DArray<FKey> keys,
        DArray<Coord> coords,
        DArray<NodeType> nodetype,
        DArray<int> joints,
        int size2d)
    {
        int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= keys.size()) return;
        ids[tId] = tId;
        keys[tId] = FKey(coords[tId][0], coords[tId][1], coords[tId][2]);
        if(tId < size2d) nodetype[tId] = NodeType::TwoD;
        else nodetype[tId] = NodeType::ThreeD;
        joints[tId] = -1;
    }

    template<typename FKey>
    __global__ void MS_SetupJoints(
        DArray<int> ids,
        DArray<FKey> keys,
		DArray<NodeType> nodetype,
        DArray<int> joints)
    {
        int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= keys.size()) return;
        if(tId != 0 && keys[tId] == keys[tId - 1])
        {
            joints[ids[tId - 1]] = ids[tId];
            joints[ids[tId]] = ids[tId - 1];
            nodetype[ids[tId - 1]] = NodeType::Joint;
            nodetype[ids[tId]] = NodeType::Joint;
        }
    }

    template<typename TDataType>
    void MixSet<TDataType>::getJointVer()
    {
    
        uint coordSize = m_coords.size();

        m_joints.resize(coordSize);
        m_verType.resize(coordSize);    

		DArray<int> coordIds;
        DArray<FKey> coordKeys;

        coordIds.resize(coordSize);
        coordKeys.resize(coordSize);

        cuExecute(coordSize,
			MS_SetupKeys,
			coordIds,
            coordKeys,
            m_coords,
            m_verType,
            m_joints,
            m_triPointSize);

        thrust::sort_by_key(thrust::device, coordKeys.begin(), coordKeys.begin() + coordKeys.size(), coordIds.begin());

        cuExecute(coordSize,
			MS_SetupJoints,
			coordIds,
            coordKeys,
			m_verType,
            m_joints);
    }

    template<typename TDataType>
    void MixSet<TDataType>::copyFrom(MixSet<TDataType> mixSet) 
    {
        if (m_coords.size() != mixSet.m_coords.size())
		{
			m_coords.resize(mixSet.m_coords.size());
		}
		m_coords.assign(mixSet.m_coords);

        m_joints.resize(mixSet.m_joints.size());
        m_joints.assign(mixSet.m_joints);

        m_verType.resize(mixSet.m_verType.size());
        m_verType.assign(mixSet.m_verType);     

        m_triangles.resize(mixSet.m_triangles.size());
        m_triangles.assign(mixSet.m_triangles);   

        m_tethedrons.resize(mixSet.m_tethedrons.size());
        m_tethedrons.assign(mixSet.m_tethedrons);   

        m_tetPointSize = mixSet.m_tetPointSize;
        m_triPointSize = mixSet.m_triPointSize;
    }

    DEFINE_CLASS(MixSet);
}